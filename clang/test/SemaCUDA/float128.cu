
#include <hip/hip_runtime.h>
// CPU-side compilation on x86 (no errors expected).
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -aux-triple nvptx64 -x cuda -fsyntax-only -verify %s

// GPU-side compilation on x86 (no errors expected)
// RUN: %clang_cc1 -triple nvptx64 -aux-triple x86_64-unknown-linux-gnu -fcuda-is-device -x cuda -fsyntax-only -verify %s

// expected-no-diagnostics
typedef _Complex float __cfloat128 __attribute__ ((__mode__ (__TC__)));
typedef __float128 _Float128;
